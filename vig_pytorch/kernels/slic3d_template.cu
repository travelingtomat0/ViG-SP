#include "hip/hip_runtime.h"
/*
Indexing:
idx = pixel/voxel index in cartesian coordinates
cidx = center index in cartesian coordinates

linear_idx = pixel/voxel index in flat array
linear_cidx = center index in flat array

Center Stride:
c_stride = number_of_features + image_dimention
center_addr = linear_cidx * c_stride

Image Stride:
z_stride = image_shape.y * image_shape.x
y_stride = image_shape.x
x_stride = 1


Transformations 3D:
linear_idx = idx.z * z_stride + idx.y * y_stride + idx.x
pixel_addr = linear_idx * n_features

idx.z = linear_idx / z_stride
plane_idx = linear_idx % z_stride
idx.y = plane_idx / y_stride
idx.x = plane_idx % y_stride

Transformations 2D:
linear_idx = idx.y * y_stride + idx.x
pixel_addr = linear_idx * n_features

idx.y = linear_idx / y_stride
idx.x = linear_idx % y_stride

*/

#define DLIMIT 99999999
#define N_FEATURES {{ n_features }}
#define N_CLUSTERS {{ n_clusters }}
#define SS {{ SS }}f

#define __min(a, b) (((a) < (b)) ? (a) : (b))
#define __max(a, b) (((a) >= (b)) ? (a) : (b))


#define sp_grid_x {{ sp_grid[0] }}
#define sp_grid_y {{ sp_grid[1] }}
#define sp_grid_z {{ sp_grid[2] }}

#define sp_shape_x {{ sp_shape[0] }}
#define sp_shape_y {{ sp_shape[1] }}
#define sp_shape_z {{ sp_shape[2] }}

#define im_shape_x {{ im_shape[0] }}
#define im_shape_y {{ im_shape[1] }}
#define im_shape_z {{ im_shape[2] }}

#define spacing_x {{ spacing[0] }}
#define spacing_y {{ spacing[1] }}
#define spacing_z {{ spacing[2] }}


__device__
float slic_distance(const int3 idx,
                    const float* pixel,
                    const long center_addr, const float* centers
)

{
    // Color diff
    float color_diff = 0;
    for ( int w = 0; w < N_FEATURES; w++ ) {
        float d = pixel[w] - centers[center_addr + w];
        color_diff += d * d;
    }

    // Position diff
    float3 pd;
    pd.z = (idx.z - centers[center_addr + N_FEATURES + 0]) * spacing_z;
    pd.y = (idx.y - centers[center_addr + N_FEATURES + 1]) * spacing_y;
    pd.x = (idx.x - centers[center_addr + N_FEATURES + 2]) * spacing_x;

    float position_diff = pd.z * pd.z + pd.y * pd.y + pd.x * pd.x;
    float dist = color_diff  +
                 position_diff / (SS);
    return dist;
}


__global__
void init_clusters(const float* data,
                   float* centers
)

{
    const long linear_cidx = threadIdx.x + (blockIdx.x * blockDim.x);

    if ( linear_cidx >= N_CLUSTERS ) {
        return;
    }

    // calculating the (0,0,0) index of each superpixel block
    // using linear to cartesian index transformation
    int3 cidx;
    int plane_size = sp_grid_y * sp_grid_x;
    cidx.z = linear_cidx / plane_size;
    int plane_idx = linear_cidx % plane_size;
    cidx.y = plane_idx / sp_grid_x;
    cidx.x = plane_idx % sp_grid_x;

    // centering index into middle of suprepixel block
    cidx.z = cidx.z * sp_shape_z + sp_shape_z / 2;
    cidx.y = cidx.y * sp_shape_y + sp_shape_y / 2;
    cidx.x = cidx.x * sp_shape_x + sp_shape_x / 2;

    //saving cluster center positions
    // note: the color is not initialized, but is kept at zero.
    const int c_stride = N_FEATURES + 3;
    centers[linear_cidx * c_stride + N_FEATURES + 0] = cidx.z;
    centers[linear_cidx * c_stride + N_FEATURES + 1] = cidx.y;
    centers[linear_cidx * c_stride + N_FEATURES + 2] = cidx.x;
}


__global__
void expectation(const float* data,
                 const float* centers,
                 unsigned int* labels
)

{
    
    int3 idx;
    idx.z = threadIdx.x + (blockIdx.x * blockDim.x);
    idx.y = threadIdx.y + (blockIdx.y * blockDim.y);
    idx.x = threadIdx.z + (blockIdx.z * blockDim.z);

    if ( idx.x >= im_shape_x ||
         idx.y >= im_shape_y ||
         idx.z >= im_shape_z ) {
        return;
    }

    long z_stride = im_shape_x * im_shape_y;
    long y_stride = im_shape_x;

    const long linear_idx = idx.z * z_stride + idx.y * y_stride + idx.x;
    const long pixel_addr = linear_idx * N_FEATURES;

    float pixel[N_FEATURES];
    for ( int w = 0; w < N_FEATURES; w++ ) {
        pixel[w] = data[pixel_addr + w];
    }

    int4 cidx, iter_cidx;
    long iter_linear_cidx;
    long closest_linear_cidx = 0;

    // approx center grid positoin
    cidx.z = __max(0, __min(idx.z / sp_shape_z, sp_grid_z - 1));
    cidx.y = __max(0, __min(idx.y / sp_shape_y, sp_grid_y - 1));
    cidx.x = __max(0, __min(idx.x / sp_shape_x, sp_grid_x - 1));

    const int c_stride = N_FEATURES + 3;
    float minimum_distance = DLIMIT;
    const int R = 2;
    for ( int k = -R; k <= R; k++ ) {
        for ( int j = -R; j <= R; j++ ) {
            for ( int i = -R; i <= R; i++ ) {
                iter_cidx.z = cidx.z + k;
                iter_cidx.y = cidx.y + j;
                iter_cidx.x = cidx.x + i;

                if ( iter_cidx.y < 0 || iter_cidx.y >= sp_grid_y || 
                     iter_cidx.z < 0 || iter_cidx.z >= sp_grid_z ||
                     iter_cidx.x < 0 || iter_cidx.x >= sp_grid_x ) {continue;}

                iter_linear_cidx = iter_cidx.z * sp_grid_y * sp_grid_x +
                                   iter_cidx.y * sp_grid_x +
                                   iter_cidx.x;
                long iter_center_addr = iter_linear_cidx * c_stride;

                if ( centers[iter_center_addr] == DLIMIT ) {
                    continue;
                }

                float dist = slic_distance(idx, pixel,
                                           iter_center_addr, centers);

                // Wrapup
                if ( dist < minimum_distance ) {
                    minimum_distance = dist;
                    closest_linear_cidx = iter_linear_cidx;
                }
            }
        }
    }

    labels[linear_idx] = closest_linear_cidx + 1;
}


__global__
void maximization(const float* data,
                  const unsigned int* labels,
                  float* centers
)

{
    const long linear_cidx = threadIdx.x + (blockIdx.x * blockDim.x);
    const int c_stride = N_FEATURES + 3;
    const long center_addr = linear_cidx * c_stride;

    if ( linear_cidx >= N_CLUSTERS ) { return; }

    int3 cidx;
    cidx.z = (int) centers[center_addr + N_FEATURES + 0];
    cidx.y = (int) centers[center_addr + N_FEATURES + 1];
    cidx.x = (int) centers[center_addr + N_FEATURES + 2];

    float ratio = 2.0f;

    int3 from;
    from.z = __max(cidx.z - sp_shape_z * ratio, 0);
    from.y = __max(cidx.y - sp_shape_y * ratio, 0);
    from.x = __max(cidx.x - sp_shape_x * ratio, 0);

    int3 to;
    to.z = __min(cidx.z + sp_shape_z * ratio, im_shape_z);
    to.y = __min(cidx.y + sp_shape_y * ratio, im_shape_y);
    to.x = __min(cidx.x + sp_shape_x * ratio, im_shape_x);


    float f[c_stride];
    for ( int k = 0; k < c_stride; k++ ) {f[k] = 0;}

    long z_stride = im_shape_x * im_shape_y;
    long y_stride = im_shape_x;

    long count = 0;
    int3 p;
    for ( p.z = from.z; p.z < to.z; p.z++ ) {
        for ( p.y = from.y; p.y < to.y; p.y++ ) {
            for ( p.x = from.x; p.x < to.x; p.x++ ) {
                long linear_idx = p.z * z_stride + p.y * y_stride + p.x;
                long pixel_addr = linear_idx * N_FEATURES;

                if ( labels[linear_idx] == linear_cidx + 1 ) {
                    for ( int w = 0; w < N_FEATURES; w++ ) {
                        f[w] += data[pixel_addr + w];
                    }
                    f[N_FEATURES + 0] += p.z;
                    f[N_FEATURES + 1] += p.y;
                    f[N_FEATURES + 2] += p.x;

                    count += 1;
                }
            }
        }
    }

    if ( count > 0 ) {
        for ( int w = 0; w < c_stride; w++ ) {
            centers[center_addr + w] = f[w] / count;
        }
    } else {
        centers[center_addr] = DLIMIT;
    }
}